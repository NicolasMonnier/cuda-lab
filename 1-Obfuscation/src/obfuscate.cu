#include "hip/hip_runtime.h"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "obfuscate.hpp"

/************************************************************************************
                                        EXERCISE 1
*************************************************************************************/

__global__
void k_cuda_exercise_1(uint8_t *d_in_buffer_1, uint8_t *d_in_buffer_2, uint32_t width, uint32_t height, uint8_t *d_out_buffer)
{
    //TODO 6: add an element coresponding to the thread index (hint: use threadIdx.x and threadIdx.y)
}

void cuda_exercise_1(uint8_t *in_buffer_1, uint8_t *in_buffer_2, uint32_t width, uint32_t height, uint8_t *out_buffer)
{
    uint8_t *d_in_buffer_1 = NULL, *d_in_buffer_2 = NULL, *d_out_buffer = NULL;
    const int size_bytes = width * height * sizeof(uint8_t);

    hipEventRecord(start_memory);
    //TODO 1: allocate memory for d_in_buffer_2 and d_out_buffer
    CUDA_CHECK(hipMalloc(&d_in_buffer_1, size_bytes));

    //TODO 2: copy in_buffer_2 to device
    hipMemcpy(d_in_buffer_1, in_buffer_1, size_bytes, hipMemcpyHostToDevice);

    hipEventRecord(start_kernel);
    //TODO 3: launch kernels with the right block dimensions
    dim3 blockSize(1, 1);
    k_cuda_exercise_1<<<1, blockSize>>>(d_in_buffer_1, d_in_buffer_2, width, height, d_out_buffer);

    hipEventRecord(start_copyback);
    //TODO 4: copy d_out_buffer back to host

    hipEventRecord(end);
    //TODO 5: don't forget to free all the device buffers
    hipFree(d_in_buffer_1);

    hipEventSynchronize(end);
}

/************************************************************************************
                                        EXERCISE 2
*************************************************************************************/

__global__
void k_cuda_exercise_2(uint32_t *d_in_buffer_1, uint32_t *d_in_buffer_2, uint32_t width, uint32_t height, uint32_t *d_out_buffer)
{
    //TODO 6: add an element coresponding to the thread index (hint: use blockIdx.x, blockIdx.y, threadIdx.x and threadIdx.y)
}

void cuda_exercise_2(uint32_t *in_buffer_1, uint32_t *in_buffer_2, uint32_t width, uint32_t height, uint32_t *out_buffer)
{
    uint32_t *d_in_buffer_1 = NULL, *d_in_buffer_2 = NULL, *d_out_buffer = NULL;
    const int size_bytes = width * height * sizeof(uint32_t);

    hipEventRecord(start_memory);
    //TODO 1: allocate memory for d_in_buffer_2 and d_out_buffer
    CUDA_CHECK(hipMalloc(&d_in_buffer_1, size_bytes));

    //TODO 2: copy in_buffer_2 to device
    hipMemcpy(d_in_buffer_1, in_buffer_1, size_bytes, hipMemcpyHostToDevice);

    hipEventRecord(start_kernel);
    //TODO 3: launch kernels with the right block and grid dimensions
    dim3 gridSize(1, 1);
    dim3 blockSize(1, 1);
    k_cuda_exercise_2<<<1, blockSize>>>(d_in_buffer_1, d_in_buffer_2, width, height, d_out_buffer);

    hipEventRecord(start_copyback);
    //TODO 4: copy d_out_buffer back to host

    hipEventRecord(end);
    //TODO 5: don't forget to free all the device buffers
    hipFree(d_in_buffer_1);

    hipEventSynchronize(end);
}

