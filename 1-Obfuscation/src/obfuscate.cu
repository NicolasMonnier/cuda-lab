#include "hip/hip_runtime.h"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "obfuscate.hpp"

/************************************************************************************
                                        EXERCISE 1
*************************************************************************************/

__global__
void k_cuda_exercise_1(uint8_t *d_in_buffer_1, uint8_t *d_in_buffer_2, uint32_t width, uint32_t height, uint8_t *d_out_buffer)
{
    //TODO 6: add an element coresponding to the thread index (hint: use threadIdx.x and threadIdx.y)
}

void cuda_exercise_1(uint8_t *in_buffer_1, uint8_t *in_buffer_2, uint32_t width, uint32_t height, uint8_t *out_buffer)
{
    uint8_t *d_in_buffer_1 = nullptr, *d_in_buffer_2 = nullptr, *d_out_buffer = nullptr;
    const int size_bytes = width * height * sizeof(uint8_t);

    CUDA_CHECK(hipMalloc(&d_in_buffer_1, size_bytes));
    //TODO 1: allocate memory for d_in_buffer_2 and d_out_buffer

    hipMemcpy(d_in_buffer_1, in_buffer_1, size_bytes, hipMemcpyHostToDevice);
    //TODO 2: copy in_buffer_2 to device

    //TODO 3: launch kernels with the right block dimensions
    dim3 blockSize(1, 1);
    k_cuda_exercise_1<<<1, blockSize>>>(d_in_buffer_1, d_in_buffer_2, width, height, d_out_buffer);

    //TODO 4: copy d_out_buffer back to host

    //TODO 5: don't forget to free all the device buffers
}

/************************************************************************************
                                        EXERCISE 2
*************************************************************************************/

__global__
void k_cuda_exercise_2(uint32_t *d_in_buffer_1, uint32_t *d_in_buffer_2, uint32_t width, uint32_t height, uint32_t *d_out_buffer)
{
    //TODO 6: add an element coresponding to the thread index (hint: use threadIdx.x and threadIdx.y)
}

void cuda_exercise_2(uint32_t *in_buffer_1, uint32_t *in_buffer_2, uint32_t width, uint32_t height, uint32_t *out_buffer)
{
    uint32_t *d_in_buffer_1 = nullptr, *d_in_buffer_2 = nullptr, *d_out_buffer = nullptr;
    const int size_bytes = width * height * sizeof(uint32_t);

    CUDA_CHECK(hipMalloc(&d_in_buffer_1, size_bytes));
    //TODO 1: allocate memory for d_in_buffer_2 and d_out_buffer

    hipMemcpy(d_in_buffer_1, in_buffer_1, size_bytes, hipMemcpyHostToDevice);
    //TODO 2: copy in_buffer_2 to device

    //TODO 3: launch kernels with the right block dimensions
    dim3 blockSize(1, 1);
    k_cuda_exercise_2<<<1, blockSize>>>(d_in_buffer_1, d_in_buffer_2, width, height, d_out_buffer);

    //TODO 4: copy d_out_buffer back to host

    //TODO 5: don't forget to free all the device buffers
}

